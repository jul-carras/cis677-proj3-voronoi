#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 1024
 
__global__ void cu_calc_dist(v_point *pixels_d, v_point *centers_d, int arraySize, long centers_size)
{
    int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int j = 0;
	long x_point, y_point, x_center, y_center;
	float distance, shortest_dist;
	int closest_points[arraySize];
	
	if(i < arraySize){
		x_point = pixels_d[i].position / dim;
		y_point = pixels_d[i].position % dim;
		
		for(j = 0; j < centers_size; j++){
			x_center = centers_d[j].position / dim;
			y_center = centers_d[j].position % dim;	
			distance = sqrt(pow(1.0 * x_center - x_point, 2) + pow(1.0 * y_center - y_point, 2));
			
			if(j == 0){	
				shortest_dist = distance;
				closest_center.position = center_list[i].position;
				closest_center.red = center_list[i].red;
				closest_center.blue = center_list[i].blue;
				closest_center.green = center_list[i].green;
			
				// if not, then check to see if the new distance we calculated is smaller
				// note this produces a first calculated point for contested areas
			} else if(distance < shortest_dist){
				shortest_dist = distance;
				closest_center.position = center_list[i].position;
				closest_center.red = center_list[i].red;
				closest_center.blue = center_list[i].blue;
				closest_center.green = center_list[i].green;
			} 
			
			pixels_d[i] = closest_center;
		}
	}
}

// This function is called from the host computer.
// It manages memory and calls the function that is executed on the GPU
extern "C" void calc_distance(v_point *pixels, v_point *centers, long array_size, long centers_size)
{
	// build GPU counterpart for each array on host, plus another to be written to
	v_point *pixels_d;
	v_point *centers_d;
	hipError_t result;

	// allocate space in the device 
	result = hipMalloc ((void**) &pixels_d, sizeof(v_point) * arraySize);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc - 'pixels' failed.");
		exit(1);		
	}
	
	result = hipMalloc ((void**) &centers_d, sizeof(v_point) * arraySize);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc - 'centers' failed.");
		exit(1);
	}

	//copy the array from host to *_d in the device 
	result = hipMemcpy (pixels_d, pixels, sizeof(v_point) * arraySize, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy - 'pixels' failed.");
		exit(1);
	}
	
	result = hipMemcpy (centers_d, centers, sizeof(v_point) * arraySize, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy - 'centers' failed.");
		exit(1);
	}

	// set execution configuration
	dim3 dimblock (BLOCK_SIZE);
	dim3 dimgrid (ceil((float) array_size/BLOCK_SIZE));

	// actual computation: Call the kernel
	cu_calc_dist <<<dimgrid, dimblock>>> (pixels_d, centers_d, array_size, centers_size);

	// transfer results back to host
	result = hipMemcpy (pixels, pixels_d, sizeof(v_point) * arraySize, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy copy to host failed.");
		exit(1);
	}
	
	// release the memory on the GPU 
	hipFree(pixels_d);
	hipFree(centers_d);
}